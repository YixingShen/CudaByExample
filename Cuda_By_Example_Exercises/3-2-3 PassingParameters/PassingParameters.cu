#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

//Declare method with __global to indicate
//function should be compiled to run on a device
__global__ void add(int a, int b, int *c) 
{
	//Add 2 numbers together and store in location pointed by *c
	*c = a + b;
}

int main(void)
{
	//Declare variables for holding data
	int c; int a = 100; int b = 50;
	int *dev_c;

	/*
	 Allocate memory on the device(GPU)
	 param one is a pointer to a pointer you want to hold the address
	 and param 2 is the size of the memory allocation.

	 Hanlde Error is a utility macro to detect any errors and exit application.
	*/
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

	//call the add method, passing parameters
	add<<<1,1>>>(a, b, dev_c);

	//Copy the memory from the device to the host so the data can be used by the host
	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

	//Print the result
	printf("%d + %d = %d\n", a, b, c);

	//Free the memory
	hipFree(dev_c);

	return 0;
}