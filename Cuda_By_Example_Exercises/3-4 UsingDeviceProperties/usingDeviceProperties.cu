//#include "../common/book.h"
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string.h>

/*This example finds a cuda device with compute capabilty 1.3 or higher*/

int main(void) 
{
	//Set a cuda device prop
	hipDeviceProp_t prop;
	int dev;

	//Get id of current CUDA device
	hipGetDevice(&dev);
	printf("ID of current CUDA device: %d\n", dev);

	//Set cuda device properties to what we need
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 3;

	//Choose a device closest to prop
	hipChooseDevice(&dev, &prop);
	printf("ID of CUDA device closest to revision 1.3: %d\n", dev);
	hipSetDevice(dev);
}