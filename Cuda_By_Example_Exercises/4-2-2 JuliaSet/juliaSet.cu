#include "hip/hip_runtime.h"
//#include "../common/book.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string.h>
#include "../common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex
{
	float r;
	float i;
	__device__ hipComplex(float a, float b) : r(a), i(b) { }
	
	//__device__ means the method will only run on the device and
	//must be called from another method running on the device.
	__device__ float magnitude2(void)
	{
		return r * r + i * i;
	}

	__device__ hipComplex operator* (const hipComplex& a)
	{
		return hipComplex(r * a.r - i * a.i, i  *a.r + r * a.i);
	}

	__device__ hipComplex operator+ (const hipComplex& a)
	{
		return hipComplex(r + a.r, i + a.i);
	}
};

__device__ int julia(int x, int y)
{
	const float scale = -1.0;
	float jx = scale * (float)(DIM/2 - x) / (DIM/2);
	float jy = scale * (float)(DIM/2 - y) / (DIM/2);

	hipComplex c(-0.8, 0.154);
	hipComplex a(jx, jy);

	int i = 0;
	for (i=0; i < 200; i++)
	{
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}

__global__ void kernal(unsigned char *ptr)
{
	//map from blockidx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;

	int offset = x + y * gridDim.x;

	// now calculate the value at that position
	int juliaValue = julia(x, y);
	ptr[offset*4 + 0] = 255 * juliaValue;
	ptr[offset*4 + 1] = 0;
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}

int main(void)
{
	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;

	hipMalloc((void**)&dev_bitmap, bitmap.image_size());

	dim3 grid(DIM, DIM);
	kernal<<<grid,1>>>(dev_bitmap);

	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
	bitmap.display_and_exit();

	hipFree(dev_bitmap);
}
