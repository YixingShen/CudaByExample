//#include "../common/book.h"
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string.h>

#define N 10

__global__ void add(int *a, int *b, int *c)
{
	int tid = blockIdx.x; // handle the data at this index
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

int main(void) 
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	//allocate the memory on the GPU
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	//Fill the arrays a and b on the CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	//Copy the arrays a and b to the GPU
	hipMemcpy(dev_a, a, N *sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N *sizeof(int), hipMemcpyHostToDevice);

	add<<<N,1>>>(dev_a, dev_b, dev_c);

	//Copy c back from the GPU to CPU
	hipMemcpy(c, dev_c, N *sizeof(int), hipMemcpyDeviceToHost);

	//Display results
	for (int i = 0; i < N; i++)
		printf("%d + %d = %d\n", a[i], b[i], c[i]);

	//Free the memory allocated to the GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}